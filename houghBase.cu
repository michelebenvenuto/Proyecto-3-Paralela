#include "hip/hip_runtime.h"
/*
 ============================================================================
 Original Author  : G. Barlas
 Added Authors    : Saúl Contreras
                  : Michele Benvenuto
 Version          : 1.1
 Last modified    : May 2022
 License          : Released under the GNU GPL 3.0
 Description      :
 To build use     : make
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "common/pgm.h"

const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;
//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
  //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  *acc = new int[rBins * degreeBins];            //el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
  memset (*acc, 0, sizeof (int) * rBins * degreeBins); //init en ceros
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  for (int i = 0; i < w; i++) //por cada pixel
    for (int j = 0; j < h; j++) //...
      {
        int idx = j * w + i;
        if (pic[idx] > 0) //si pasa thresh, entonces lo marca
          {
            int xCoord = i - xCent;
            int yCoord = yCent - j;  // y-coord has to be reversed
            float theta = 0;         // actual angle
            for (int tIdx = 0; tIdx < degreeBins; tIdx++) //add 1 to all lines in that pixel
              {
                float r = xCoord * cos (theta) + yCoord * sin (theta);
                int rIdx = (r + rMax) / rScale;
                (*acc)[rIdx * degreeBins + tIdx]++; //+1 para este radio r y este theta
                theta += radInc;
              }
          }
      }
}

//*****************************************************************
// Uso de memoria constante para la tabla de senos y cosenos
// inicializarlo en main y pasarlo al device
__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];

//*****************************************************************
//TODO Kernel memoria compartida
__global__ void GPU_HoughTranShared(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale)
{
  int i;
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  if (gloID > w * h) return;      // in case of extra threads

  int locID = threadIdx.x;
  int xCent = w / 2;
  int yCent = h / 2;

  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  __shared__ int localAcc[degreeBins * rBins];

  for (i = locID; i < degreeBins * rBins; i += blockDim.x)
    localAcc[i] = 0;
  
  // esperar a que todos los threads hagan la inicializacion
  __syncthreads();

  if (pic[gloID] > 0)
    {
      for (int tIdx = 0; tIdx < degreeBins; tIdx ++)
        {
          float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
          int rIdx = (r + rMax) / rScale;
          atomicAdd (localAcc + (rIdx * degreeBins + tIdx), 1);
        }
    }
  
    // esepramos a que todos los warps terminen los calculos
    __syncthreads();

    // atomic add para acumular lo local en lo global
    for (i = locID; i < degreeBins * rBins; i+= blockDim.x)
      atomicAdd (acc + i, localAcc[i]);
}

//TODO Kernel memoria Constante
__global__ void GPU_HoughTranConst(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale)
{
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  if (gloID > w * h) return;      // in case of extra threads in block

  int xCent = w / 2;
  int yCent = h / 2;

  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  if (pic[gloID] > 0)
    {
      for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
          //TODO utilizar memoria constante para senos y cosenos
          float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx]; //probar con esto para ver diferencia en tiempo
          int rIdx = (r + rMax) / rScale;
          // Debemos usar atomic, debido a que en los pixeles
          // más cercanos al punto central se puede generar
          // race conditions debido al concepto de línea
          // en cuanto a gráficas por computador se refiere.
          // ya que estos son valores discretos y no continuos
          atomicAdd (acc + (rIdx * degreeBins + tIdx), 1);
        }
    }
}

// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
__global__ void GPU_HoughTran (unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float *d_Cos, float *d_Sin)
{
  //TODO calcular: int gloID = ?
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  if (gloID > w * h) return;      // in case of extra threads in block

  int xCent = w / 2;
  int yCent = h / 2;

  //TODO explicar bien bien esta parte. Dibujar un rectangulo a modo de imagen sirve para visualizarlo mejor
  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  //TODO eventualmente usar memoria compartida para el acumulador

  if (pic[gloID] > 0)
    {
      for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
          //TODO utilizar memoria constante para senos y cosenos
          //float r = xCoord * cos(tIdx) + yCoord * sin(tIdx); //probar con esto para ver diferencia en tiempo
          float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
          int rIdx = (r + rMax) / rScale;
          //debemos usar atomic, pero que race condition hay si somos un thread por pixel? explique
          atomicAdd (acc + (rIdx * degreeBins + tIdx), 1);
        }
    }

  //TODO eventualmente cuando se tenga memoria compartida, copiar del local al global
  //utilizar operaciones atomicas para seguridad
  //faltara sincronizar los hilos del bloque en algunos lados

}

//*****************************************************************
int main (int argc, char **argv)
{
  int i;

  PGMImage inImg (argv[1]);

  int *cpuht;
  int w = inImg.x_dim;
  int h = inImg.y_dim;

  //Variables utilizadas para medir el tiempo de ejecucion con cuda events
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMalloc ((void **) &d_Cos, sizeof (float) * degreeBins);
  hipMalloc ((void **) &d_Sin, sizeof (float) * degreeBins);

  // CPU calculation
  CPU_HoughTran(inImg.pixels, w, h, &cpuht);

  // pre-compute values to be stored
  float *pcCos = (float *) malloc (sizeof (float) * degreeBins);
  float *pcSin = (float *) malloc (sizeof (float) * degreeBins);
  float rad = 0;
  for (i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos (rad);
    pcSin[i] = sin (rad);
    rad += radInc;
  }

  float rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  // TODO eventualmente volver memoria global

  // CONSTANT MEMORIES CHANES
  //
  //
  //
  hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof (float) * degreeBins);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof (float) * degreeBins);
  //
  //
  //
  //

  // setup and copy data from host to device
  unsigned char *d_in, *h_in;
  int *d_hough, *h_hough;

  h_in = inImg.pixels; // h_in contiene los pixeles de la imagen

  h_hough = (int *) malloc (degreeBins * rBins * sizeof (int));

  hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
  hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
  hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

  // execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
  //1 thread por pixel
  int blockNum = ceil (w * h / 256);

  hipEventRecord(start);
  GPU_HoughTranShared <<< blockNum, 256 >>> (d_in, w, h, d_hough, rMax, rScale);
  hipEventRecord(stop);
  hipDeviceSynchronize (); 
  
  // get results from device
  hipMemcpy (h_hough, d_hough, sizeof (int) * degreeBins * rBins, hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);

  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);

  // compare CPU and GPU results
  for (i = 0; i < degreeBins * rBins; i++)
  {
    if (cpuht[i] != h_hough[i])
    printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
  }
  printf("Done!\n");
  
  printf("Execution time: %f milliseconds\n", ms);
  
  // TODO clean-up
  hipFree ((void *) d_Cos);
  hipFree ((void *) d_Sin);
  hipFree ((void *) d_in);
  hipFree ((void *) d_hough);
  free (h_hough);
  free (cpuht);
  free (pcCos);
  free (pcSin);
  hipDeviceReset ();

  return 0;
}
